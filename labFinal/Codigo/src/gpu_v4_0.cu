#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>

#ifndef _UCHAR
#define _UCHAR
    typedef unsigned char uchar;
#endif

/* 
 * V4.0 (Bucket sort):
 * - Cada hilo carga en memoria compartida la entrada de la matriz correspondiente.
 * - Cada hilo ordena su propia ventana.
 * - Cambia el cálculo de mediana:
 *   Se utiliza Bucket Sort para contar las ocurrencias de valores y se calcula la mediana a partir de ellos sin ordenar el arreglo.
*/

__device__ __forceinline__
uchar bucketMedian(uchar *window, int size) {
    // Se usa uchar porque se asume que ninguna ventana va a tener 2^8 = 256 pixeles del mismo color (soporta hasta ventanas de 16*16)
    uchar buckets[256] = {0}; // inicializa en 0, magia de P2

    for (int i = 0; i < size ; i++)
        buckets[ window[i] ]++;

    int cont = 0;
    int i = 0;

    for (i = 0; (i < 256) && (cont <= size / 2); i++)
        cont += buckets[i];

    return i-1;
}

__global__ 
void filtro_mediana_kernel_v4_0(uchar* d_input, uchar* d_output, int width, int height, int W, float threshold) {

    extern __shared__ uchar sharedMem[];

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    int radius = W / 2;

    // Cargo el bloque en memoria compartida
    int sharedIndex = threadIdx.x + threadIdx.y * blockDim.x;

    sharedMem[sharedIndex] = d_input[x + y * width];
    __syncthreads();
    
    // Coordenadas del bloque
    int blockStartX = blockIdx.x * blockDim.x;
    int blockStartY = blockIdx.y * blockDim.y;

    // Variables auxiliares del cálculo de ventana
    int idx = 0;
    int currentX;
    int currentY;

    // window en shared
    uchar* window = &sharedMem[blockDim.x * blockDim.y + sharedIndex * W * W];

    for (int dy = -radius; dy <= radius; dy++) {
        for (int dx = -radius; dx <= radius; dx++) {

            currentX = x+dx;
            currentY = y+dy;
             
            if(currentX >= 0 && currentY >= 0 && currentX < width && currentY < height){ // Si estoy dentro de la matriz:
                window[idx++] = (currentX >= blockStartX) && 
                                (currentX < blockStartX + blockDim.x) && 
                                (currentY >= blockStartY) && 
                                (currentY < blockStartY + blockDim.y) 
                                ? sharedMem[(threadIdx.x+dx) + (threadIdx.y+dy) * blockDim.x] : d_input[currentX + currentY * width]; // Leo desde shared si estoy dentro del bloque, o desde global sino
            } else  // Si no estoy dentro de la matriz
                window[idx++] = 0;
        }
    }

    // Ordeno los valores de la ventana
    uchar median = bucketMedian(window, idx); // En idx quedó cargado el tamaño de la ventana

    d_output[y * width + x] = median;

}

void filtro_mediana_gpu_v4_0(uchar* img_in, uchar* img_out, int width, int height, int W, int blockSize) {

    uchar *d_in, *d_out;
    size_t size = width * height * sizeof(uchar);

    hipMalloc(&d_in, size);
    hipMalloc(&d_out, size);

    // Copia de datos desde el host al dispositivo
    hipMemcpy(d_in, img_in, size, hipMemcpyHostToDevice);

    dim3 blockDim(blockSize, blockSize);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);
    
    // Tamaño de memoria compartida
    size_t sharedMemSize = (blockDim.x * blockDim.y) * (1 + W * W) * sizeof(uchar);

    // Lanzamiento del kernel
    filtro_mediana_kernel_v4_0<<<gridDim, blockDim, sharedMemSize>>>(d_in, d_out, width, height, W, 0);
    
    hipMemcpy(img_out, d_out, size, hipMemcpyDeviceToHost);
    
    // Liberación de memoria del dispositivo
    hipFree(d_in);
    hipFree(d_out);
}