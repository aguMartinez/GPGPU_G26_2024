#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <algorithm>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "nvToolsExt.h"
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/transform.h>
#include <thrust/scatter.h>
#include <thrust/copy.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>

__global__ void filtro_mediana_kernel(float* d_input, float* d_output, int width, int height, int W, float threshold) {
    extern __shared__ float sharedMem[];

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    int radius = W / 2;

    int blockWidth = blockDim.x + 2 * radius;
    int blockHeight = blockDim.y + 2 * radius;

    int localX = threadIdx.x + radius;
    int localY = threadIdx.y + radius;

    // Cargo la ventana en memoria compartida
    for (int dy = -radius; dy <= radius; dy++) {
        for (int dx = -radius; dx <= radius; dx++) {
            int nx = min(max(x + dx, 0), width - 1);
            int ny = min(max(y + dy, 0), height - 1);

            sharedMem[(localY + dy) * blockWidth + (localX + dx)] = d_input[ny * width + nx];
        }
    }

    __syncthreads();

    // Extraigo los valores de la ventana
    float window[15 * 15]; // Usa un tamaño estático suficiente para la ventana más grande posible
    int idx = 0;

    for (int dy = -radius; dy <= radius; dy++) {
        for (int dx = -radius; dx <= radius; dx++) {
            window[idx++] = sharedMem[(localY + dy) * blockWidth + (localX + dx)];
        }
    }

    // Ordeno los valores de la ventana
    for (int i = 0; i < idx - 1; i++) {
        for (int j = 0; j < idx - i - 1; j++) {
            if (window[j] > window[j + 1]) {
                float temp = window[j];
                window[j] = window[j + 1];
                window[j + 1] = temp;
            }
        }
    }

    float median = window[idx / 2];

    if (fabs(d_input[y * width + x] - median) > threshold) {
        d_output[y * width + x] = median;
    } else {
        d_output[y * width + x] = d_input[y * width + x];
    }
}

void filtro_mediana_gpu(float* img_in, float* img_out, int width, int height, int W) {
    printf("Filtro mediana GPU %d \n", W);
    float* d_in, *d_out;
    size_t size = width * height * sizeof(float);
    hipMalloc(&d_in, size);
    hipMalloc(&d_out, size);

    hipMemcpy(d_in, img_in, size, hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);

    size_t sharedMemSize = (blockDim.x + W - 1) * (blockDim.y + W - 1) * sizeof(float);

    filtro_mediana_kernel<<<gridDim, blockDim, sharedMemSize>>>(d_in, d_out, width, height, W, 0);

    hipDeviceSynchronize(); // Sincronización para asegurarse de que todos los kernels hayan terminado

    hipMemcpy(img_out, d_out, size, hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);
    printf("Filtro mediana GPU %d terminado\n", W);
}

void filtro_mediana_cpu(float * img_in, float * img_out, int width, int height, int W) {
    int radius = W / 2;

    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            std::vector<float> neighborhood;

            for (int dy = -radius; dy <= radius; dy++) {
                for (int dx = -radius; dx <= radius; dx++) {
                    int nx = x + dx;
                    int ny = y + dy;
                    if (nx >= 0 && nx < width && ny >= 0 && ny < height) {
                        neighborhood.push_back(img_in[ny * width + nx]);
                    }
                }
            }

            std::nth_element(neighborhood.begin(), neighborhood.begin() + neighborhood.size() / 2, neighborhood.end());
            float median = neighborhood[neighborhood.size() / 2];

            int threshold = 0;
            // printf("Median: %f\n", fabs(img_in[y * width + x] - median));
            
            if (fabs(img_in[y * width + x] - median) > threshold) {
                img_out[y * width + x] = median;
            } else {
                img_out[y * width + x] = img_in[y * width + x];
            }
            
        }
    }
}
