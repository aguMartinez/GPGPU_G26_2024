#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <algorithm>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "nvToolsExt.h"

__global__ void filtro_mediana_kernel(float* d_input, float* d_output, int width, int height, int W, float threshold) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    int radius = W / 2;
    int max_neighborhood_size = W * W;
    float* neighborhood = (float*)malloc(max_neighborhood_size * sizeof(float));
    int idx = 0;

    for (int dy = -radius; dy <= radius; dy++) {
        for (int dx = -radius; dx <= radius; dx++) {
            int nx = x + dx;
            int ny = y + dy;
            if (nx >= 0 && nx < width && ny >= 0 && ny < height) {
                neighborhood[idx++] = d_input[ny * width + nx];
            }
        }
    }

    for (int i = 0; i < idx - 1; i++) {
        for (int j = 0; j < idx - i - 1; j++) {
            if (neighborhood[j] > neighborhood[j + 1]) {
                float temp = neighborhood[j];
                neighborhood[j] = neighborhood[j + 1];
                neighborhood[j + 1] = temp;
            }
        }
    }

    float median = neighborhood[idx / 2];

    if (fabs(d_input[y * width + x] - median) > threshold) {
        d_output[y * width + x] = median;
    } else {
        d_output[y * width + x] = d_input[y * width + x];
    }
}


void filtro_mediana_gpu(float* img_in, float* img_out, int width, int height, int W) {
    float* d_in, *d_out;
    size_t size = width * height * sizeof(float);
    hipMalloc(&d_in, size);
    hipMalloc(&d_out, size);

    hipMemcpy(d_in, img_in, size, hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);

    filtro_mediana_kernel<<<gridDim, blockDim>>>(d_in, d_out, width, height, W, 0);

    hipMemcpy(img_out, d_out, size, hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);
}
