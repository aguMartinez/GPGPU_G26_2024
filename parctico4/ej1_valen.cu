/* Ejercicio 1 | Practico 4 | Grupo 26 | GPGPU 2024*/

#include <stdlib.h>
#include <stdio.h>
#include "hip/hip_runtime.h"

#define TILE_FILA 16
#define TILE_COLUMNA 32

#define CUDA_CHK(ans)                         \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

int *randomMatrix(int f, int c)
{
    int *A = (int *)malloc(f * c * sizeof(int));

    for (int i = 0; i < f * c; i++)
    {
        A[i] = rand() % 100;
    }
    return A;
}

int isTransponse(int * m1, int * m2, int c, int f) {

    for (int i = 0; i < f; i++) {
        for (int j = 0; j < c; j ++) {
            if (m1[i * c + j] != m2[j*f + i]) {
                return 0;
            }
        }
    }
    return 1;
}

void printMatrix(int *A, int f, int c)
{
    for (int i = 0; i < f; i++)
    {
        for (int j = 0; j < c; j++)
        {
            printf("%d ", A[i * c + j]);
        }
        printf("\n");
    }
}

__global__ void transposeKernel(int *d_M, int *d_MTrans, int f, int c)
{
    __shared__ int tile[TILE_COLUMNA * TILE_FILA];

    int x = blockIdx.x * TILE_COLUMNA + threadIdx.x;
    int y = blockIdx.y * TILE_FILA + threadIdx.y;
    // Cargar en tile de manera transpuesta

    if (x < c && y < f)
        tile[threadIdx.x * TILE_FILA + threadIdx.y] = d_M[y * c + x];

    __syncthreads();

    int num_pos = threadIdx.y * TILE_COLUMNA + threadIdx.x;

    int offset = blockIdx.x * TILE_COLUMNA * f + blockIdx.y * TILE_FILA ;

    int cuantoMePasoColumna = num_pos / TILE_FILA;
    int cuantoMePasoFila = num_pos % TILE_FILA;

    int posTile = cuantoMePasoColumna * f + cuantoMePasoFila;

    if (num_pos < TILE_COLUMNA * TILE_FILA && offset + posTile < f*c)
        d_MTrans[offset + posTile] = tile[num_pos];
}

    /*
    int a_transponer = threadIdx.y * TILE_COLUMNA + threadIdx.x;
    int k = a_transponer / TILE_COLUMNA;
    int l = a_transponer % TILE_COLUMNA;
    */

    // Se invierten los indices de los bloques, pues es es el bloque transpuesto.
    /*
    int x_trans = blockIdx.y * TILE_COLUMNA + threadIdx.x;
    int y_trans = blockIdx.x * TILE_FILA + threadIdx.y;

    int n_esimo_elemento = threadIdx.y * TILE_COLUMNA + threadIdx.x;
    int k = n_esimo_elemento / TILE_COLUMNA;
    int l = n_esimo_elemento % TILE_COLUMNA;

    if (j < f && i < c)  // Ajustar los limites para la matriz transpuesta
    {
        // Escribir por fila en la memoria global

        d_MTrans[y_trans * c + x_trans] = tile[l * TILE_FILA + k];
    }
    */

int main(int argc, char *argv[])
{
    if (argc != 3)
    {
        printf("Faltaron argumentos %d \n", argc);
        return 1;
    }

    /* definir tamanios de matriz*/
    int f = atoi(argv[1]);
    int c = atoi(argv[2]);

    for (int input = 1; input < 3; input++)
    {
        printf("%d ", atoi(argv[input]));
    }

    printf("\n");

    int size = f * c * sizeof(int);

    /* generar matrices del host */
    int *h_M = randomMatrix(f, c);
    int *h_MTrans = (int *)malloc(size);

    /* reservar memoria en la GPU */
    int *d_M;
    int *d_MTrans;
    CUDA_CHK(hipMalloc((void **)&d_M, size));      // matriz de entrada
    CUDA_CHK(hipMalloc((void **)&d_MTrans, size)); // matriz de salida

    /* copiar los datos de entrada a la GPU */
    hipMemcpy(d_M, h_M, size, hipMemcpyHostToDevice);

    /* Ej 1:*/
    dim3 threadsPerBlock(TILE_COLUMNA, TILE_FILA);
    dim3 numBlocks((c + TILE_COLUMNA - 1) / TILE_COLUMNA, (f + TILE_FILA - 1) / TILE_FILA);
    printf("numblocks: %d %d\n", numBlocks.x, numBlocks.y);

    for (int i = 0; i < 10; i++)
    {
        transposeKernel<<<numBlocks, threadsPerBlock>>>(d_M, d_MTrans, f, c);
    }

    /* Copiar los datos de salida a la CPU en h_message */
    hipMemcpy(h_MTrans, d_MTrans, size, hipMemcpyDeviceToHost);

    /* Imprimir resultados*/

    printf("trapureeesuhkfdkk %d \n", isTransponse(h_M, h_MTrans, c, f));

    printMatrix(h_M, f, c);
    printf("-----------------------\n");
    printMatrix(h_MTrans, c, f);

    /* Liberar memoria */
    free(h_M);
    free(h_MTrans);
    hipFree(d_M);
    hipFree(d_MTrans);

    return 1;
}
