/* Ejercicio 3 | Practico 4 | Grupo 26 | GPGPU 2024 */

#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"

#define HISTOGRAM_LENGTH 8
#define FILAS 16
#define COLUMNAS 16

#define CUDA_CHK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int *randomMatrix(int f, int c)
{
    int *A = (int *)malloc(f * c * sizeof(int));

    for (int i = 0; i < f * c; i++)
    {
        A[i] = rand() % HISTOGRAM_LENGTH;
    }
    return A;
}

void printMatrix(int *A, int f, int c)
{
    for (int i = 0; i < f; i++)
    {
        for (int j = 0; j < c; j++)
        {
            printf("%d ", A[i * c + j]);
        }
        printf("\n");
    }
}

__global__ void decrypt_kernel_ej3B(int *d_M, int * d_H, int * d_MH)
{
	 __shared__ int shared_histogram[HISTOGRAM_LENGTH];
	
	int x = blockIdx.x * blockDim.x + threadIdx.x; 
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	int num_pos = threadIdx.y * blockDim.x + threadIdx.x;

	if (num_pos < HISTOGRAM_LENGTH) {
		shared_histogram[num_pos] = 0;
	}

	__syncthreads();

	int elem = d_M[y * COLUMNAS + x];
	//shared_histogram[elem]++;
	atomicAdd(&shared_histogram[elem], 1);
	
	__syncthreads();

	int block_pos = blockIdx.y * gridDim.x + blockIdx.x;

	if (num_pos < HISTOGRAM_LENGTH) {
		d_MH[block_pos * HISTOGRAM_LENGTH + num_pos] = shared_histogram[num_pos];
	}
}

__global__ void reduction(int* d_MH, int numRows, int numCols, int salto) {
	extern __shared__ int intermedio[];

	int blockDimX = blockDim.x;
	
	int x = blockIdx.x * blockDimX + threadIdx.x; 
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	int num_pos = threadIdx.y * blockDimX + threadIdx.x;

	intermedio[num_pos] = d_MH[(y+salto) * HISTOGRAM_LENGTH + x];

	__syncthreads();

	int i = blockDimX * blockDim.y / 2;
	while (i >= blockDimX) {
		if (num_pos < i) {
			//printf("(%d,%d) (%d,%d) %d + %d \n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, intermedio[num_pos], intermedio[num_pos+i]);
			intermedio[num_pos] = intermedio[num_pos] + intermedio[num_pos+i];
		}
		__syncthreads();
		i = i / 2;
	}
    __syncthreads();

	// Guardar los resultado
	if (num_pos < blockDimX) {
		d_MH[y *  HISTOGRAM_LENGTH + x] = intermedio[num_pos];
	}
}

int main(int argc, char *argv[])
{

	if (argc != 3)
    {
        printf("Faltaron argumentos %d \n", argc);
        return 1;
    }

    /* definir tamanios de matriz*/
    int blockX = atoi(argv[1]);
    int blockY = atoi(argv[2]);

    for (int input = 1; input < 3; input++)
    {
        printf("%d ", atoi(argv[input]));
    }

    printf("\n");

	int sizeH = HISTOGRAM_LENGTH * sizeof(int);
	int sizeM = FILAS * COLUMNAS * sizeof(int);

	int* h_M = randomMatrix(FILAS, COLUMNAS);
	int* h_H = (int*) malloc(sizeH);

	memset(h_H, 0, sizeH);

	int* d_M; 
	int* d_H;

	/* reservar memoria en la GPU */
	CUDA_CHK(hipMalloc((void **)&d_H, sizeH));
	CUDA_CHK(hipMalloc((void **)&d_M, sizeM));

	/* copiar los datos de entrada a la GPU */

 	hipMemcpy(d_M, h_M, sizeM, hipMemcpyHostToDevice);
 	hipMemcpy(d_H, h_H, sizeH, hipMemcpyHostToDevice);

	dim3 threadsPerBlock(blockX, blockY);
	dim3 numBlocks( (COLUMNAS + blockX - 1) / blockX, (FILAS + blockY - 1) / blockY);

	int sizeMH = numBlocks.x * numBlocks.y * HISTOGRAM_LENGTH * sizeof(int);

	int* h_MH = (int*) malloc(sizeMH);
	int* d_MH;

	CUDA_CHK(hipMalloc((void **)&d_MH, sizeMH));

    decrypt_kernel_ej3B<<<numBlocks, threadsPerBlock>>>(d_M, d_H, d_MH);

	/* Copiar los datos de salida a la CPU en h_message */
	// cudaMemcpy(h_H, d_H, sizeH, cudaMemcpyDeviceToHost);
	hipMemcpy(h_MH, d_MH, sizeMH, hipMemcpyDeviceToHost);

	printf("-----------MATRIZ ORIGINAL------------\n");
	printMatrix(h_M, FILAS, COLUMNAS);

	int remainingRows = numBlocks.x * numBlocks.y;
	hipMemcpy(h_MH, d_MH, sizeMH, hipMemcpyDeviceToHost);
	printf("-----------MATRIZ HISTOGRAMA ANTES DEL REDUCE------------\n");
	printMatrix(h_MH, remainingRows, HISTOGRAM_LENGTH);

	
	//begin initial_reduction
	remainingRows = numBlocks.x * numBlocks.y;  //Todas las filas restantes
	blockX = 4; //Pasar por input
	blockY = 4; //Pasar por input

	dim3 threadsPerBlock2(blockX, blockY);
	dim3 numBlocks2((HISTOGRAM_LENGTH + blockX - 1) / blockX, (remainingRows + blockY - 1) / blockY);

	int sizeShared = blockX * blockY * sizeof(int);
	reduction<<<numBlocks2, threadsPerBlock2, sizeShared>>>(d_MH, remainingRows, HISTOGRAM_LENGTH, 0);
	//end initial_reduction
	/*
	//begin loop
	remainingRows = numBlocks.x;
	blockY = 4; //Idealmente se pasa por input
	blockX = 1024/blockY;
	sizeShared = blockX * blockY * sizeof(int);
	dim3 threadsPerBlock3(blockX, blockY);
	dim3 numBlocks3((HISTOGRAM_LENGTH + blockX - 1) / blockX, (remainingRows + blockY - 1) / blockY);

	reduction<<<numBlocks2, threadsPerBlock2, sizeShared>>>(d_MH, remainingRows, HISTOGRAM_LENGTH, remainingRows);

	//end loop when remainingRows = 0
	*/
	

 	hipMemcpy(h_MH, d_MH, sizeMH, hipMemcpyDeviceToHost);

    printf("-----------MATRIZ HISTOGRAMA LUEGO DEL REDUCE------------\n");
	printMatrix(h_MH, remainingRows, HISTOGRAM_LENGTH);

	// libero la memoria en la CPU
	free(h_M);
	free(h_H);
	free(h_MH);

	// libero la memoria en la GPU
	hipFree(d_M);
    hipFree(d_H);
	hipFree(d_MH);

	return 0;
}
