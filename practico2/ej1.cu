#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"

#define CUDA_CHK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void read_file(const char*, int*);
int get_text_length(const char * fname);

#define A 15
#define B 27
#define M 256
#define A_MMI_M -17

#define N 512


__device__ int modulo(int a, int b){
	int r = a % b;
	r = (r < 0) ? r + b : r;
	return r;
}

__global__ void decrypt_kernel(int *d_message, int length)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < length) {
		d_message[i] = modulo(A_MMI_M*(d_message[i] - B), M);
	}
}

void ej1A(int* d_message, int length) {
	int threads_per_block = 256;
	int blocks_per_grid = 1;

	decrypt_kernel<<<blocks_per_grid, threads_per_block>>>(d_message, length);
}

void ej1B(int* d_message, int length) {
	int threads_per_block = 256;
	int blocks_per_grid = (length + threads_per_block - 1) / threads_per_block;

	decrypt_kernel<<<blocks_per_grid, threads_per_block>>>(d_message, length);
}

void ej1C(int* d_message, int length) {
	int blocks_per_grid = 1024;
	int threads_per_block = (length + blocks_per_grid - 1) / blocks_per_grid;
	printf("%d, %d",threads_per_block, length);

	decrypt_kernel<<<blocks_per_grid, threads_per_block>>>(d_message, length);
}

int main(int argc, char *argv[])
{
	int *h_message;
	int *d_message;
	unsigned int size;

	const char * fname;

	if (argc < 2) printf("Debe ingresar el nombre del archivo\n");
	else
		fname = argv[1];

	int length = get_text_length(fname);

	size = length * sizeof(int);

	// reservar memoria para el mensaje
	h_message = (int *)malloc(size);

	// leo el archivo de la entrada
	read_file(fname, h_message);

	/* reservar memoria en la GPU */
	CUDA_CHK(hipMalloc((void **)&d_message, size));

	/* copiar los datos de entrada a la GPU */

 	hipMemcpy(d_message, h_message, size, hipMemcpyHostToDevice);

	/* Configurar la grilla y lanzar el kernel */
	/* Descomentar la parte de ejercicio a ejecutar */
	//ej1A(d_message, length);
	//ej1B(d_message, length);
	ej1C(d_message, length);

	/* Copiar los datos de salida a la CPU en h_message */
 	hipMemcpy(h_message, d_message, size, hipMemcpyDeviceToHost);

	// despliego el mensaje
	for (int i = 0; i < length; i++) {
		printf("%c", (char)h_message[i]);
	}
	printf("\n");
	

	// libero la memoria en la GPU
	hipFree(d_message);

	// libero la memoria en la CPU
	free(h_message);

	return 0;
}


int get_text_length(const char * fname)
{
	FILE *f = NULL;
	f = fopen(fname, "r"); //read and binary flags

	size_t pos = ftell(f);    
	fseek(f, 0, SEEK_END);    
	size_t length = ftell(f); 
	fseek(f, pos, SEEK_SET);  

	fclose(f);

	return length;
}

void read_file(const char * fname, int* input)
{
	// printf("leyendo archivo %s\n", fname );

	FILE *f = NULL;
	f = fopen(fname, "r"); //read and binary flags
	if (f == NULL){
		fprintf(stderr, "Error: Could not find %s file \n", fname);
		exit(1);
	}

	//fread(input, 1, N, f);
	int c; 
	while ((c = getc(f)) != EOF) {
		*(input++) = c;
	}

	fclose(f);
}
