#include "hip/hip_runtime.h"
/* Ejercicio 1 | Practico 3 | Grupo 26 | GPGPU 2024*/

#include <stdlib.h>
#include <stdio.h>
#include "hip/hip_runtime.h"

#define CUDA_CHK(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int* randomMatrix(int n, int m){
    int* A = (int*) malloc(n * m *sizeof(int*));

    for (int i = 0; i < n * m; i++) {
        A[i] = rand();
    }
    return A;
}

__global__ void transpose_kernel(int* d_M, int* d_MTrans, int n){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y

    if (x < n && y < n){
        d_MTrans[y + x * n] = d_M[x + y * n];
    }
}


 int main(){

    /* definir tamanios de matriz*/
    int n = 256;
    int m = n;
    int size = n*m*sizeof(int)
    
    int* R;

    /* generar matriz*/
    int* h_M;
    h_M = randomMatrix(n,m);

    /* reservar memoria en la GPU */
    int* d_M;
    int* d_MTrans;
	CUDA_CHK(hipMalloc((void **)&d_M, size)); //matriz de entrada
    CUDA_CHK(hipMalloc((void **)&d_MTrans, size)); //matriz de salida

	/* copiar los datos de entrada a la GPU */

 	hipMemcpy(d_M, h_M, size, hipMemcpyHostToDevice);

    /* Ej 1:*/
	int threads_per_block = 32*32;
    int blocks_per_grid = 128; //Todo: arreglar este num.

    transpose_kernel<<<blocks_per_grid, threads_per_block>>>(d_M, d_MTrans size);
    
    /* Copiar los datos de salida a la CPU en h_message */
 	hipMemcpy(h_MTrans, d_MTrans, size, hipMemcpyDeviceToHost);
    return 1;
 }