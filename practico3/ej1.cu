/* Ejercicio 1 | Practico 3 | Grupo 26 | GPGPU 2024*/

#include <stdlib.h>
#include <stdio.h>
#include "hip/hip_runtime.h"

#define CUDA_CHK(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int* randomMatrix(int n, int m){
    int* A = (int*) malloc(n * m *sizeof(int));

    for (int i = 0; i < n * m; i++) {
        A[i] = rand();
    }
    return A;
}

void printMatrix(int* A, int n, int m){
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            printf("%d ", A[i * m + j]);
        }
        printf("\n");
    }
}

__global__ void transposeKernel(int* d_M, int* d_MTrans, int n, int m){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < n && y < m){
        d_MTrans[y + x * n] = d_M[x + y * m];
    }
}


 int main(){

    /* definir tamanios de matriz*/
    int n = 64;
    int m = n;
    int size = n*m*sizeof(int);
    
    /* generar matrices del host */
    int* h_M = randomMatrix(n,m);
    int* h_MTrans = (int*)malloc(size);

    /* reservar memoria en la GPU */
    int* d_M;
    int* d_MTrans;
	CUDA_CHK(hipMalloc((void **)&d_M, size)); //matriz de entrada
    CUDA_CHK(hipMalloc((void **)&d_MTrans, size)); //matriz de salida

	/* copiar los datos de entrada a la GPU */
 	hipMemcpy(d_M, h_M, size, hipMemcpyHostToDevice);

    /* Ej 1:*/
	dim3 threadsPerBlock(32,32);
    dim3 numBlocks((m + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (n + threadsPerBlock.y - 1) / threadsPerBlock.y);
    transposeKernel<<<numBlocks, threadsPerBlock>>>(d_M, d_MTrans, n,m);
    
    /* Copiar los datos de salida a la CPU en h_message */
 	hipMemcpy(h_MTrans, d_MTrans, size, hipMemcpyDeviceToHost);

    /* Imprimir resultados*/
    printMatrix(h_M, n, m);
    printf("-----------------------\n");
    printMatrix(h_MTrans, n, m);

    /* Liberar memoria */
    free(h_M);
    free(h_MTrans);
    hipFree(d_M);
    hipFree(d_MTrans);

    return 1;
}

