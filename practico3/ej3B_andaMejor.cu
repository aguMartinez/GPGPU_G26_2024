/* Ejercicio 3 | Practico 3 | Grupo 26 | GPGPU 2024*/

#include <stdlib.h>
#include <stdio.h>
#include "hip/hip_runtime.h"

#define CUDA_CHK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define N_COLUMNS 10

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int* randomMatrix(int m, int n){
    int* A = (int*) malloc(m * n *sizeof(int));

    for (int i = 0; i < m * n; i++){
        A[i] = rand()%100;
    }
    return A;
}

void printMatrix(int* A, int m, int n){
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++){
            printf("%d ", A[i * n + j]);
        }
        printf("\n");
    }
}

__global__ void optimizedMatrixVectorKernel(int* A, int* v, int* Av, int numRows, int numCols) {
    __shared__ int sdata[1];

    int row = blockIdx.x;  // Cada bloque maneja una fila completa
    int col = threadIdx.x; // Cada hilo en el bloque maneja una columna

    if (threadIdx.x == 0) {
        sdata[0] = 0;  // Inicializa el acumulador en memoria compartida
    }

    __syncthreads();

    atomicAdd(&sdata[0], A[row * numCols + col] * v[col]);

    __syncthreads();

    if (threadIdx.x == 0) {
        Av[row] = sdata[0];
    }

}


int main(){

    /* definir tamanios de matriz*/
    int m = 10;
    int n = N_COLUMNS;

    int inputMatrixSize = m*n*sizeof(int);
    int outputSize = m*1*sizeof(int);
    int vectorSize = n*1*sizeof(int);

    /* generar matriz, vector y matriz resultado*/
    int* h_M = randomMatrix(m,n);
    int* h_v = randomMatrix(n,1);
    int* h_Mv = (int*)malloc(outputSize);
    memset(h_Mv, 0, outputSize);

    /* reservar memoria en la GPU */
    int *d_M;
    int *d_v;
    int *d_Mv;
	CUDA_CHK(hipMalloc((void **)&d_M, inputMatrixSize)); //matriz de entrada
    CUDA_CHK(hipMalloc((void **)&d_v, vectorSize)); //vector
    CUDA_CHK(hipMalloc((void **)&d_Mv, outputSize)); //matriz de salida

	/* copiar los datos de entrada a la GPU */
 	hipMemcpy(d_M, h_M, inputMatrixSize, hipMemcpyHostToDevice);
    hipMemcpy(d_Mv, h_Mv, outputSize, hipMemcpyHostToDevice);
    hipMemcpy(d_v, h_v, vectorSize, hipMemcpyHostToDevice);
/*
    dim3 threadsPerBlock(1, n);
    dim3 numBlocks(m / threadsPerBlock.x,
                   n / threadsPerBlock.y);

*/

    dim3 threadsPerBlock(n);
    dim3 numBlocks(m);

    for (int i = 0; i < 10; i ++) {
        optimizedMatrixVectorKernel<<<numBlocks, threadsPerBlock>>>(d_M, d_v, d_Mv, m, n);
    }

    // Copiar los datos del vector de salida de la GPU al host
    hipMemcpy(h_Mv, d_Mv, outputSize, hipMemcpyDeviceToHost);


    printMatrix(h_M, m,n);
    printf("--------------\n");
    printMatrix(h_v, n,1);
    printf("--------------\n");
    printMatrix(h_Mv, m,1);

    /* Liberar memoria */
    free(h_M);
    free(h_v);
    free(h_Mv);
    hipFree(d_M);
    hipFree(d_v);
    hipFree(d_Mv);

    return 1;
 }