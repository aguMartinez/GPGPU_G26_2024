/* Ejercicio 2 | Practico 3 | Grupo 26 | GPGPU 2024 */

#include <stdlib.h>
#include <stdio.h>
#include "hip/hip_runtime.h"

#define CUDA_CHK(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int* randomMatrix(int n, int m){
    int* A = (int*) malloc(n * m *sizeof(int));

    for (int i = 0; i < n * m; i++) {
        A[i] = rand() % 100;
    }
    return A;
}

void printMatrix(int* A, int n, int m){
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            printf("%d ", A[i * m + j]);
        }
        printf("\n");
    }
}

__global__ void sum4thPosition_kernel(int* d_M, int* d_MRes, int n, int m){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (y+4 < n && y < m){
        d_MRes[x + y * n] = d_M[x + y * n] + d_M[x + (y+4) * n];
    }
}

int main(int argc, char* argv[]){

    if (argc != 5) {
        printf("Faltaron argumentos %d \n", argc);
        return 1;
    }

    /* definir tamanios de matriz*/
    int m = atoi(argv[1]);
    int n = atoi(argv[2]);

    int blockX = atoi(argv[3]);
    int blockY = atoi(argv[4]);

    for(int input=0; input<5; input++){
        printf("%d ",atoi(argv[input]));
    }
    
    printf("\n");
    int size = n*m*sizeof(int);

    /* generar matriz*/
    int* h_M = randomMatrix(n,m);
    int* h_MRes = (int*)malloc(size);

    /* reservar memoria en la GPU */
    int* d_M;
    int* d_MRes;
	CUDA_CHK(hipMalloc((void **)&d_M, size)); //matriz de entrada
    CUDA_CHK(hipMalloc((void **)&d_MRes, size)); //matriz de salida

	/* copiar los datos de entrada a la GPU */
 	hipMemcpy(d_M, h_M, size, hipMemcpyHostToDevice);

    /* Ej 2a */
	dim3 threadsPerBlock(blockX,blockY);
    dim3 numBlocks(m / threadsPerBlock.x,
                   n / threadsPerBlock.y);

    for(int i = 0; i < 10; i++){
        sum4thPosition_kernel<<<numBlocks, threadsPerBlock>>>(d_M, d_MRes, n, m);
    }
    
    /* Copiar los datos de salida a la CPU */
 	hipMemcpy(h_MRes, d_MRes, size, hipMemcpyDeviceToHost);

    /* Imprimir resultados*/
    /*
    printMatrix(h_M, n, m);
    printf("-----------------------\n");
    printMatrix(h_MRes, n, m);
    */
    
    /* Liberar memoria */
    free(h_M);
    free(h_MRes);
    hipFree(d_M);
    hipFree(d_MRes);

    return 1;
 }