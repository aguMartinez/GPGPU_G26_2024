/* Ejercicio 1 | Practico 4 | Grupo 26 | GPGPU 2024*/

#include <stdlib.h>
#include <stdio.h>
#include "hip/hip_runtime.h"

#define CUDA_CHK(ans)                         \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

int *randomMatrix(int f, int c)
{
    int *A = (int *)malloc(f * c * sizeof(int));

    for (int i = 0; i < f * c; i++)
    {
        A[i] = i;
    }
    return A;
}

int matrixIsTransposed(int * m1, int * m2, int c, int f) {

    for (int i = 0; i < f; i++) {
        for (int j = 0; j < c; j ++) {
            if (m1[i * c + j] != m2[j*f + i]) {
                return 0;
            }
        }
    }
    return 1;
}

void printMatrix(int *A, int f, int c)
{
    for (int i = 0; i < f; i++)
    {
        for (int j = 0; j < c; j++)
        {
            printf("%d ", A[i * c + j]);
        }
        printf("\n");
    }
}

__global__ void transposeKernel(int *d_M, int *d_MTrans, int f, int c){
    extern __shared__ int tile[];

    int global_x = blockIdx.x * blockDim.x + threadIdx.x;
    int global_y = blockIdx.y * blockDim.y + threadIdx.y;
    

    tile[threadIdx.x * blockDim.y + threadIdx.y] = d_M[(global_y) * c + (blockIdx.x * blockDim.x + threadIdx.x)];

    __syncthreads();

    int num_pos = threadIdx.y * blockDim.x + threadIdx.x;

    int offset = blockIdx.x * blockDim.x * f + blockIdx.y * blockDim.y;

    int posTile = (num_pos / blockDim.y * f) + (num_pos % blockDim.y);

    d_MTrans[offset + posTile] = tile[num_pos];
}

__global__ void transposeKernelDummy(int *d_M, int *d_MTrans, int f, int c){
    extern __shared__ int tile[];

    tile[threadIdx.x * (blockDim.y+1) + threadIdx.y] = d_M[(blockIdx.y * blockDim.y + threadIdx.y) * c + (blockIdx.x * blockDim.x + threadIdx.x)];

    __syncthreads();

    //num_pos identifica al n-esimo thread del bloque
    int num_pos_tile = threadIdx.y * blockDim.x + threadIdx.x + threadIdx.y;
    int num_pos = threadIdx.y * blockDim.x + threadIdx.x;

    //offset es la posicion del elemento (0,0) del  bloque a trapsoner en la matriz resultado
    int offset = blockIdx.x * blockDim.x * f + blockIdx.y * blockDim.y;
 
    int posTile = (num_pos / blockDim.y * f) + (num_pos % blockDim.y);

    int pos_dM = offset + posTile;


    d_MTrans[pos_dM] = tile[num_pos_tile];
}


int main(int argc, char *argv[])
{
    if (argc < 5)
    {
        printf("Faltaron argumentos %d \n", argc);
        return 1;
    }

    /* definir tamanios de matriz*/
    int f = atoi(argv[1]);
    int c = atoi(argv[2]);

    int blockX = atoi(argv[3]);
    int blockY = atoi(argv[4]);

    for (int input = 1; input < 5; input++)
    {
        printf("%d ", atoi(argv[input]));
    }

    printf("\n");

    int size = f * c * sizeof(int);

    /* generar matrices del host */
    int* h_M = randomMatrix(f, c);
    int* h_MTrans = (int *)malloc(size);

    /* reservar memoria en la GPU */
    int *d_M;
    int *d_MTrans;
    CUDA_CHK(hipMalloc((void **)&d_M, size));      // matriz de entrada
    CUDA_CHK(hipMalloc((void **)&d_MTrans, size)); // matriz de salida

    /* copiar los datos de entrada a la GPU */
    hipMemcpy(d_M, h_M, size, hipMemcpyHostToDevice);

    /* Ej 1a:*/
    dim3 threadsPerBlock(blockX, blockY);
    dim3 numBlocks((c + blockX - 1) / blockX, (f + blockY - 1) / blockY);

    int sizeTile = blockX * blockY * sizeof(int);

    for (int i = 0; i < 10; i++)
    {
        transposeKernel<<<numBlocks, threadsPerBlock, sizeTile>>>(d_M, d_MTrans, f, c);
    }

    /* Copiar los datos de salida a la CPU en h_message */
    hipMemcpy(h_MTrans, d_MTrans, size, hipMemcpyDeviceToHost);

    /* Imprimir resultados*/

    printf("transpuesta: %d \n",matrixIsTransposed(h_M,h_MTrans,c,f));

    free(h_MTrans);
    hipFree(d_MTrans);

    /*Ej 1b:*/

    h_MTrans = (int *)malloc(size);
    CUDA_CHK(hipMalloc((void **)&d_MTrans, size));


    int sizeTileDummy = (blockX) * (blockY+1) * sizeof(int);

    for (int i = 0; i < 10; i++)
    {
        transposeKernelDummy<<<numBlocks, threadsPerBlock, sizeTileDummy>>>(d_M, d_MTrans, f, c);
    }

    /* Copiar los datos de salida a la CPU en h_message */
    hipMemcpy(h_MTrans, d_MTrans, size, hipMemcpyDeviceToHost);
    


    /* Imprimir resultados*/
    printf("transpuesta Dummy: %d ",matrixIsTransposed(h_M,h_MTrans,c,f));
    

    /* Liberar memoria */
    free(h_M);
    free(h_MTrans);
    hipFree(d_M);
    hipFree(d_MTrans);

    return 1;
}
