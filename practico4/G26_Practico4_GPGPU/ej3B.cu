/* Ejercicio 3 | Practico 4 | Grupo 26 | GPGPU 2024 */

#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include <math.h>

#define HISTOGRAM_LENGTH 256
#define FILAS 2160
#define COLUMNAS 3840

#define CUDA_CHK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int *randomMatrix(int f, int c, int multiplo)
{
    int filas_extra = (multiplo - (f % multiplo)) % multiplo;
    int *A = (int *)malloc((f + filas_extra) * c * sizeof(int));

    for (int i = 0; i < f * c; i++)
    {
        A[i] = 	i % HISTOGRAM_LENGTH;
    }

    for (int i = f * c; i < (filas_extra + f) * c; i++)
    {
        A[i] = 0;
    }

    return A;
}

/*Imprime filas salteadas de la matriz A*/
void printMatrixSalto(int* A, int f, int c, int salto) {
    for (int i = 0; i < f; i += salto) {
        for (int j = 0; j < c; j++) {
            printf("%d ", A[i * c + j]);
        }
        printf("\n");
    }
}

void printMatrix(int* A, int f, int c)
{
    for (int i = 0; i < f; i++)
    {
        for (int j = 0; j < c; j++)
        {
            printf("%d ", A[i * c + j]);
        }
        printf("\n");
    }
}

__global__ void decrypt_kernel_ej3B(int *d_M, int * d_MH, int cant_rows)
{
	__shared__ int shared_histogram[HISTOGRAM_LENGTH];
	
	int x = blockIdx.x * blockDim.x + threadIdx.x; 
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	int num_pos = threadIdx.y * blockDim.x + threadIdx.x;

	if (num_pos < HISTOGRAM_LENGTH) {
		shared_histogram[num_pos] = 0;
	}

	__syncthreads();

    if (x < COLUMNAS && y < cant_rows) {
        int elem = d_M[y * COLUMNAS + x];
        atomicAdd(&shared_histogram[elem], 1);
    }

	__syncthreads();

	int block_pos = blockIdx.y * gridDim.x + blockIdx.x;

	if (num_pos < HISTOGRAM_LENGTH) {
		d_MH[block_pos * HISTOGRAM_LENGTH + num_pos] = shared_histogram[num_pos];
	}
}

__global__ void reduction(int* d_MH, int numRows, int salto) {
	extern __shared__ int intermedio[];
	
	int x = blockIdx.x * blockDim.x + threadIdx.x; 
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	int num_pos = threadIdx.y * blockDim.x + threadIdx.x;

	if ((y * HISTOGRAM_LENGTH + x + y * salto * HISTOGRAM_LENGTH) < (numRows * HISTOGRAM_LENGTH)) {
		intermedio[num_pos] = d_MH[y * HISTOGRAM_LENGTH + x + y * salto * HISTOGRAM_LENGTH];
	} else {
		intermedio[num_pos] = 0;
	}

	__syncthreads();

	int i = blockDim.x * blockDim.y / 2;
	while (i >= blockDim.x) {
		if (num_pos < i) {
			intermedio[num_pos] = intermedio[num_pos] + intermedio[num_pos+i];
		}
		__syncthreads();
		i = i / 2;
	}
    __syncthreads();

	// Guardar los resultado
	if (num_pos < blockDim.x && ((y * HISTOGRAM_LENGTH + x + y * salto * HISTOGRAM_LENGTH) < (numRows * HISTOGRAM_LENGTH))) {
		d_MH[y * HISTOGRAM_LENGTH + x + y * salto * HISTOGRAM_LENGTH] = intermedio[num_pos];
	}
}

int main(int argc, char *argv[])
{
	if (argc != 5)
    {
        printf("Faltaron argumentos %d \n", argc);
        return 1;
    }

    /* definir tamanios de matriz*/
    int blockX = atoi(argv[1]);
    int blockY = atoi(argv[2]);

	int blockXReduction = atoi(argv[3]);
    int blockYReduction = atoi(argv[4]);

    for (int input = 1; input < 5; input++)
    {
        printf("%d ", atoi(argv[input]));
    }

    printf("\n");

	int* h_M = randomMatrix(FILAS, COLUMNAS, blockY);
	int* d_M; 

    int new_filas = FILAS + ((blockY - (FILAS % blockY)) % blockY);

    int sizeM = new_filas * COLUMNAS * sizeof(int);

	CUDA_CHK(hipMalloc((void **)&d_M, sizeM));

 	hipMemcpy(d_M, h_M, sizeM, hipMemcpyHostToDevice);


	dim3 threadsPerBlock(blockX, blockY);
	dim3 numBlocks( (COLUMNAS + blockX - 1) / blockX, (new_filas / blockY));

	int rowsMH = numBlocks.x * numBlocks.y;
	int sizeMH = rowsMH * HISTOGRAM_LENGTH * sizeof(int);
	int remainingRows = rowsMH;

	int* h_MH = (int*) malloc(sizeMH);
	int* d_MH;

	CUDA_CHK(hipMalloc((void **)&d_MH, sizeMH));

    decrypt_kernel_ej3B<<<numBlocks, threadsPerBlock>>>(d_M, d_MH, new_filas);
	
	/* Copiar los datos de salida a la CPU en h_message */
	hipMemcpy(h_MH, d_MH, sizeMH, hipMemcpyDeviceToHost);

	int sizeShared = blockX * blockY * sizeof(int);
	long salto = 1;
	
	threadsPerBlock.x = blockXReduction;
	threadsPerBlock.y = blockYReduction;
	numBlocks.x = (HISTOGRAM_LENGTH + blockXReduction - 1) / blockXReduction;
	numBlocks.y = (remainingRows + blockYReduction - 1) / blockYReduction;

	while (remainingRows > 1) {
		reduction<<<numBlocks, threadsPerBlock, sizeShared>>>(d_MH, rowsMH, salto - 1);
		hipDeviceSynchronize();

		salto = salto * blockYReduction;

		remainingRows = (remainingRows + blockYReduction - 1) / blockYReduction;

        numBlocks.y = remainingRows;
		hipMemcpy(h_MH, d_MH, sizeMH, hipMemcpyDeviceToHost);
	}

 	hipMemcpy(h_MH, d_MH, sizeMH, hipMemcpyDeviceToHost);

	printf("-----------MATRIZ HISTOGRAMA LUEGO DEL REDUCE------------\n");
	printMatrix(h_MH, 1, HISTOGRAM_LENGTH);
	printf("---------------------------------\n");


	// libero la memoria en la CPU
	free(h_M);
	free(h_MH);

	// libero la memoria en la GPU
	hipFree(d_M);
	hipFree(d_MH);

	return 0;
}
