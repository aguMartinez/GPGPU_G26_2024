#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "hip/hip_runtime.h"


int* constantVector(int n, int c){
  int* V = (int*)malloc(n * sizeof(int));

  for (int i = 0; i < n; i++){
      V[i] = c;
  }
  return V;
}
void printVector(int* V, int n){
  for (int i = 0; i < n; i++){
      printf("%d ", V[i]);
  }
  printf("\n");
}

__global__ void exclusiveScanKernel(int* d_out, int* d_in, int n) {
  extern __shared__ int temp[];

  int thid = threadIdx.x;
  int offset = 1;

  temp[2 * thid] = d_in[2 * thid];
  temp[2*thid+1] = d_in[2*thid+1];
  //parte 1: up-sweep
  for (int d = n/2; d > 0; d /= 2){
    if(thid < d){
      int ai = offset * (2 * thid + 1) - 1;
      int bi = offset * (2 * thid + 2) - 1;
      temp[bi] += temp[ai];
    }
    offset *= 2;
  }
  
  //ultimo elemento se pone en cero
  if (thid == 0) {
    temp[n - 1] = 0;
  }

  //down-sweep
  for (int d = 1; d < n; d *= 2){
    offset /= 2;
    __syncthreads();

    //[debug] begin
    //printf("[%d, %d] %d \n", d, thid, temp[thid]);
    //[debug] end

    if (thid < d) {
      int ai = offset * (2 * thid + 1) - 1;
      int bi = offset * (2 * thid + 2) - 1;
      int t = temp[ai];
      temp[ai] = temp[bi];
      temp[bi] += t;
    }
  }
  __syncthreads();


  d_out[2 * thid] = temp[2 * thid];
  d_out[2 * thid + 1] = temp[2 * thid + 1];
  
  //[debug] begin
  //printf("[%d] %d \n", thid, temp[thid]);
  //[debug] end

}


int main(int argc, char *argv[]) {
  
  if (argc < 2){
    printf("Faltaron argumentos %d \n", argc);
    return 1;
  }

  /* definir tamanios de matriz*/
  int n = atoi(argv[1]);
  int blockX = 1024;

  for (int input = 1; input < argc; input++)
  {
    printf("%d ", atoi(argv[input]));
  }
  printf("\n");
  /* genero vector de entrada*/

  int* h_in = constantVector(n,1);
  int* h_out = (int*)malloc(n*sizeof(int));


  int* d_in;
  int* d_out;
  size_t size = n * sizeof(int);

  

  hipMalloc(&d_in, size);
  hipMalloc(&d_out, size);

  hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);

  double interval = 0;

  int threadsPerBlock = blockX;
  int blocks = (n + threadsPerBlock - 1) / threadsPerBlock;
  size_t shared_mem_size =  n * sizeof(int);

  
  exclusiveScanKernel<<<blocks, threadsPerBlock, shared_mem_size>>>(d_out, d_in, n);



  printf("Interval: %f\n", interval);
  hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);

  
  //printVector(h_in, n);
  printVector(h_out, n);
  

  hipFree(d_in);
  hipFree(d_out);
}