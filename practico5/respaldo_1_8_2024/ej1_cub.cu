#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>

#define MS(f,elap)                                                                                           \
{                                                                                                            \
struct timespec t_ini,t_fin;                                                                                 \
    clock_gettime(CLOCK_MONOTONIC, &t_ini);                                                                  \
    f;                                                                                                       \
    clock_gettime(CLOCK_MONOTONIC, &t_fin);                                                                  \
    elap = 1000 * (t_fin.tv_sec - t_ini.tv_sec) + (t_fin.tv_nsec - t_ini.tv_nsec)/1000000.0;                 \
}

int* constantVector(int n){
  int* V = (int*)malloc(n * sizeof(int));

  for (int i = 0; i < n; i++){
      V[i] = 1;
  }
  return V;
}

void printVector(int* V, int n){
  for (int i = 0; i < n; i++){
      printf("%d ", V[i]);
  }
  printf("\n");
}




/* 
 * Implentación basada en documentación oficial:
 * https://github.com/dmlc/cub/blob/master/cub/device/device_scan.cuh
*/

int main(int argc, char *argv[]) {
  
  if (argc < 2){
    printf("Faltaron argumentos %d \n", argc);
    return 1;
  }
  
 
  int num_items = atoi(argv[1]);
  printf("num_items = %d\n", num_items);
  
  int* host_data = constantVector(num_items);
  
  //variables de dispositivo
  

  
  int* d_in;
  int* d_out;
  
  void* d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;




  hipMalloc(&d_in, num_items * sizeof(int));
  hipMalloc(&d_out, num_items * sizeof(int));
  hipMemcpy(d_in, host_data, num_items * sizeof(int), hipMemcpyHostToDevice);

  double interval = 0;
  MS(
  //primera llamada para calcular el tamaño del espacio temporal
  hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items);

  hipMalloc(&d_temp_storage, temp_storage_bytes);

  hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items);


  , interval);
  hipMemcpy(host_data, d_out, num_items * sizeof(int), hipMemcpyDeviceToHost);



  //liberar memoria
  hipFree(d_in);
  hipFree(d_out);
  hipFree(d_temp_storage);

  /*
  printVector(host_data, num_items);
  */
  printf("Interval: %f\n", interval);




  free(host_data);

  return 0;

}